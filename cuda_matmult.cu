
#include <hip/hip_runtime.h>
#include<iostream>
#define tile_width 16

using namespace std;

__global__ void Mat_Mul_Shared(float *d_A, float *d_B, float *d_C, int width) {

	 int i = blockIdx.y * blockDim.y + threadIdx.y;
	 int j = blockIdx.x * blockDim.x + threadIdx.x;

	 __shared__ float M[tile_width][tile_width];
	 __shared__ float N[tile_width][tile_width];

	 int n_tile = width / tile_width;  //number of tiles
	 float value = 0;

	 for(int m=0; m < n_tile; m++ ){  // m loop of tiles
			 // copy from globalb memory to shared memory       
			 M[threadIdx.y][threadIdx.x] = d_A[i*width + m*tile_width + threadIdx.x];
			 N[threadIdx.y][threadIdx.x] = d_B[(m*tile_width + threadIdx.y)*width + j];
			 __syncthreads();  // wait all threads finish copying

			for(int k=0; k < tile_width; k++){  // k loop within a tile, in the m loop of tiles: together loop over width to perfomr the dot product of a row of d_A  and a coloumn of d_B
					value += M[threadIdx.y][k] * N[k][threadIdx.x];  // scalar value is in the register of a thread
			}
			__syncthreads();  // wait ll threads finish partial dot product
	 }

	 d_C[i*width + j] = value;  // assign value in register of every thread to corresponding elements of d_C in global memory

}


int main(){

	int n;
	// cin >> n;
	n = 10000;
	int size = n*n*sizeof(float);

	//float *A, *B, *C;
	float *A = (float *) malloc( size );  // allocate CPU memory
	float *B = (float *) malloc( size );
	float *C = (float *) malloc( size );

	for(int i=0; i<n; i++)
		 for(int j=0; j<n; j++){
				 int idx = i*n + j;
				 A[idx] = rand()%10;
				 B[idx] = rand()%10;
				 C[idx] = 0;
	}


	float *d_A, *d_B, *d_C;

	// CUDA events for timing
    hipEvent_t start, stop, start_load, stop_load, start_unload, stop_unload;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	hipEventCreate(&start_load);
    hipEventCreate(&stop_load);
	hipEventCreate(&start_unload);
    hipEventCreate(&stop_unload);

	// Load

	hipEventRecord(start_load);  // Record start load time

	hipMalloc(&d_A, size);  // allocate GPU memory
	hipMalloc(&d_B, size);
	hipMalloc(&d_C, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

	int block_width = tile_width;
	int grid_width = ceil(n/block_width);
	// dim3 is a CUDA built-in struct
	dim3 dim_grid(grid_width, grid_width);  // num of blocks = (int(n/16) + 1)^2, efficient if > 6.
	dim3 dim_block(block_width, block_width);  // number of theads per block = 16*16 =256

	hipEventRecord(stop_load);  // Record stop load time
    hipEventSynchronize(stop_load);  // Wait for the stop load event to complete
    float milliseconds_load = 0;
    hipEventElapsedTime(&milliseconds_load, start_load, stop_load);  // Calculate elapsed time for loading
	cout << milliseconds_load << endl;

	// Execution

	hipEventRecord(start);  // Record start execution time

	Mat_Mul_Shared<<<dim_grid, dim_block>>> (d_A, d_B, d_C, n);  // lauch kenel

	hipEventRecord(stop);  // Record stop execution time
    hipEventSynchronize(stop);  // Wait for the stop execution event to complete
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);  // Calculate elapsed time for execution
	cout << milliseconds << endl;

	// Unload

	hipEventRecord(start_unload);  // Record start execution time

	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

	hipEventRecord(stop_unload);  // Record stop time
    hipEventSynchronize(stop_unload);  // Wait for the stop event to complete
    float milliseconds_unload = 0;
    hipEventElapsedTime(&milliseconds_unload, start_unload, stop_unload);  // Calculate elapsed time
	cout << milliseconds_unload << endl;

//   for(int i=0; i<n; i++)
//      for(int j=0; j<n; j++){
//      cout<<i<<"  "<<j<<"  "<<C[i*n+j]<<endl;
//   } 

	cout << "Time taken for matrix multiplication on GPU: " << endl;
	cout << "Load from CPU to GPU: " << milliseconds_load << " ms" << endl;
	cout << "Execution Time : " << milliseconds << " ms" << endl;
	cout << "Unload to CPU from GPU: " << milliseconds_unload << " ms" << endl;
	cout << "Total Time: " << milliseconds + milliseconds_load + milliseconds_unload << " ms" << endl;

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	free(A);
	free(B);
	free(C);



}
